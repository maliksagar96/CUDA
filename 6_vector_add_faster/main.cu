/*

Benchcmarked the codes for CPU and GPU.

Play with N and see what your graphic card shows you.

*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

using namespace std;

#define N 10000000

void init_vector(float *vect, int n) {
  for(int i = 0;i<n;i++) {
    vect[i] = static_cast<float>(rand())/RAND_MAX;
  }
}

double get_time() {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec + ts.tv_nsec * 1e-9;
}

void vector_add_cpu(float *a, float *b, float *c, int n) {
  for(int i = 0;i<n;i++) {
    c[i] = a[i] + b[i];
  }
}

__global__ void vector_add_gpu(float *a, float *b, float *c, int n) {

  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID < n) {
    c[threadID] = a[threadID] + b[threadID];
  }

}

int main() {

  size_t size = N*sizeof(float);

  int threads_per_block = 256;
  int num_blocks = 1 + (N-1)/threads_per_block;

  float *h_a = (float*)malloc(size);
  float *h_b = (float*)malloc(size);
  float *h_c = (float*)malloc(size);
  float *h_c_gpu = (float*)malloc(size);

  float *d_a, *d_b, *d_c;

  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  srand(time(NULL));
  init_vector(h_a, N);
  init_vector(h_b, N);

  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  printf("Performing warmup runs ...\n");
  for(int i = 0;i<3;i++) {
    vector_add_cpu(h_a, h_b, h_c, N);
    vector_add_gpu<<<num_blocks, threads_per_block>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
  }


  printf("Benchmarking CPU implementation...\n");

  double cpu_total_time = 0.0;
  for(int i = 0;i<20;i++) {
    double start_time = get_time();
    vector_add_cpu(h_a,h_b,h_c, N);
    double end_time = get_time();
    cpu_total_time += end_time - start_time;
  }
  double cpu_avg_time = cpu_total_time / 20.0;


  printf("Benchmarking GPU implementation...\n");
  double gpu_total_time = 0.0;
  for(int i = 0;i<20;i++) {
    double start_time = get_time();
    vector_add_gpu<<<num_blocks, threads_per_block>>>(d_a,d_b,d_c,N);
    hipDeviceSynchronize();
    double end_time = get_time();
    gpu_total_time += end_time - start_time;
  }
  double gpu_avg_time = gpu_total_time / 20.0;

  // Print results
  printf("CPU average time: %f milliseconds\n", cpu_avg_time*1000);
  printf("GPU average time: %f milliseconds\n", gpu_avg_time*1000);
  printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);


  hipMemcpy(h_c_gpu, d_c, size, hipMemcpyDeviceToHost);


  bool cpu_gpu_equal = true;
  for(int i = 0;i<N;i++) {
    if(fabs(h_c_gpu[i] - h_c[i]) > 1e-5) {
      cpu_gpu_equal = false;
      break;
    }
  }

  if(cpu_gpu_equal) printf("CPU and GPU calculations are equal.\n");
  else printf("CPU and GPU calculations are not equal\n");

  free(h_a);
  free(h_b);
  free(h_c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}