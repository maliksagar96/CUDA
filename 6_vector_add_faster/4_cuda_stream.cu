/*

Now this is a simple sample code to use cuda streams. 

Now think streams as multiple pipes from which water is flowing. If you have multiple pipes then you can send water through each pipe simultaneouly and 
increase the parallelism of your code. 

Here the data is divided into chunks so that it can se sent through each stream independently. 

Also the memory transfer from host to device and from device to host is aysnchronous, that means the cpu doesn't wait for the memory transfer. 
It can happen in the background and cpu can start doing other tasks and you can launch other kernels in the meantime.

*/


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 10000000

void init_vector(float *vect, int n) {
  for(int i = 0; i < n; i++) {
    vect[i] = static_cast<float>(rand()) / RAND_MAX;
  }
}

double get_time() {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec + ts.tv_nsec * 1e-9;
}

void vector_add_cpu(float *a, float *b, float *c, int n) {
  for(int i = 0; i < n; i++) {
    c[i] = a[i] + b[i];
  }
}

__global__ void vector_add_gpu(float *a, float *b, float *c, int n) {
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID < n) {
    c[threadID] = a[threadID] + b[threadID];
  }
}

int main() {
  size_t size = N * sizeof(float);

  int numstreams = 10;
  int chunk_size = N/numstreams;
  

  int threads_per_block = 256;
  int num_blocks = (N + threads_per_block - 1) / threads_per_block;

  float *h_a, *h_b, *h_c, *h_c_gpu;
  float *d_a, *d_b, *d_c;

  // Use pinned memory on host
  hipHostMalloc(&h_a, size, hipHostMallocDefault);
  hipHostMalloc(&h_b, size, hipHostMallocDefault);
  hipHostMalloc(&h_c, size, hipHostMallocDefault);
  hipHostMalloc(&h_c_gpu, size, hipHostMallocDefault);

  // Device memory
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  srand(time(NULL));
  init_vector(h_a, N);
  init_vector(h_b, N);

  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  printf("Performing warmup runs ...\n");
  for(int i = 0; i < 3; i++) {
    vector_add_cpu(h_a, h_b, h_c, N);
    vector_add_gpu<<<num_blocks, threads_per_block>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
  }

  printf("Benchmarking CPU implementation...\n");
  double cpu_total_time = 0.0;
  for(int i = 0; i < 20; i++) {
    double start_time = get_time();
    vector_add_cpu(h_a, h_b, h_c, N);
    double end_time = get_time();
    cpu_total_time += end_time - start_time;
  }
  double cpu_avg_time = cpu_total_time / 20.0;

  hipStream_t stream[numstreams];
  for(int i = 0;i<numstreams;i++) {
    hipStreamCreate(&stream[i]);
  }

printf("Benchmarking GPU implementation...\n");

float gpu_total_time = 0.0f;

hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

int stream_blocks = 1 + (chunk_size-1)/threads_per_block;

for (int i = 0; i < 20; i++) {
  hipEventRecord(start, 0);
  for(int s = 0;s<numstreams;s++) {
    int offset = s * chunk_size;
    
    hipMemcpyAsync(d_a + offset, h_a + offset, chunk_size * sizeof(float), hipMemcpyHostToDevice, stream[s]);
    hipMemcpyAsync(d_b + offset, h_b + offset, chunk_size * sizeof(float), hipMemcpyHostToDevice, stream[s]);
    vector_add_gpu<<<stream_blocks, threads_per_block, 0, stream[s]>>>(d_a + offset, d_b + offset, d_c + offset, chunk_size);
    hipMemcpyAsync(h_c_gpu + offset, d_c + offset, chunk_size * sizeof(float), hipMemcpyDeviceToHost, stream[s]);
  }

  for(int s = 0;s<numstreams;s++) {
    hipStreamSynchronize(stream[s]);
  }
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float milliseconds = 0.0f;
  hipEventElapsedTime(&milliseconds, start, stop);
  gpu_total_time += milliseconds;
}

hipEventDestroy(start);
hipEventDestroy(stop);

float gpu_avg_time = gpu_total_time / 20.0f;

printf("CPU average time: %f milliseconds\n", cpu_avg_time * 1000);
printf("GPU average time: %f milliseconds\n", gpu_avg_time);
printf("Speedup: %fx\n", (cpu_avg_time * 1000) / gpu_avg_time);

  bool cpu_gpu_equal = true;
  for(int i = 0; i < N; i++) {
    if(fabs(h_c_gpu[i] - h_c[i]) > 1e-5) {
      cpu_gpu_equal = false;
      break;
    }
  }

  if(cpu_gpu_equal)
    printf("CPU and GPU calculations are equal.\n");
  else
    printf("CPU and GPU calculations are not equal\n");

  for(int i = 0;i<numstreams;i++) {
    hipStreamDestroy(stream[i]); 
  }

  // Free pinned host memory
  hipHostFree(h_a);
  hipHostFree(h_b);
  hipHostFree(h_c);
  hipHostFree(h_c_gpu);

  // Free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
