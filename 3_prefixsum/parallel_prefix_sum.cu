#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void naive_prefix_sum(int* d_a, int* d_prefix_sum, int n){

  int i = threadIdx.x;
  if(i<n) {
    int sum = 0;
    for(int j = 0;j<=i;j++) {
      sum += d_a[j];
    }
    d_prefix_sum[i] = sum;
  }
}

int main() {
  int n = 10;
  int* a = (int*)malloc(sizeof(int) * n);
  int* prefix_sum = (int*)malloc(sizeof(int) * n);

  int *d_a,*d_prefix_sum;

  hipMalloc((void**)&d_a, sizeof(int)*n);
  hipMalloc((void**)&d_prefix_sum, sizeof(int)*n);

  for (int i = 0; i < n; i++) {
    a[i] = i;
  }

  hipMemcpy(d_a, a, sizeof(int)*n, hipMemcpyHostToDevice);

  naive_prefix_sum<<<1,n>>>(d_a, d_prefix_sum, n);
  hipMemcpy(prefix_sum, d_prefix_sum, sizeof(int)*n, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  
  for(int i = 0;i<n;i++) {
    printf("prefix sum[%d] = %d\n", i, prefix_sum[i]);
  }

  free(a);
  free(prefix_sum);
  hipFree(d_a);
  hipFree(d_prefix_sum);
}